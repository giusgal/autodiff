#include "hip/hip_runtime.h"
// test_jacobian_cuda.cpp
#include <iostream>
#include <Eigen/Dense>
#include <chrono>
#include "DualVar.hpp"
#include "ForwardDifferentiator.hpp"
#include "CudaSupport.hpp"
#include "example-functions.hpp"

using dv = autodiff::forward::DualVar<double>;
using dvec = Eigen::Matrix<dv, Eigen::Dynamic, 1>;

int main() {
    using Clock = std::chrono::high_resolution_clock;

    // Problem dimensions
    int dim_in = testfun::input_dim;   // Input dimension
    int dim_out = testfun::output_dim;  // Output dimension

    // Create input vector
    Eigen::VectorXd x0 = Eigen::VectorXd::Random(dim_in);
    Eigen::VectorXd real_eval_cpu(dim_out);

    // Create Jacobian object
    Eigen::MatrixXd j(dim_out, dim_in);
    

    // Test regular CPU compute
    auto t1 = Clock::now();
    autodiff::forward::jacobian<double>(testfun::test_fun, x0, real_eval, j);
    auto t2 = Clock::now();
    auto cpu_time = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();

    std::cout << "CPU Jacobian norm:\n" << jacobian.getJacobian().norm() << std::endl;
    std::cout << "CPU Time: " << cpu_time << " μs\n\n";

#ifdef USE_CUDA
    Eigen::MatrixXd jc(dim_out, dim_in);
    Eigen::VectorXd real_eval(dim_out);

    autodiff::forward::CudaFunctionWrapper<double> cudafun = testfun::createcudafn();
    // Test CUDA compute
    auto t3 = Clock::now();
    autodiff::forward::jacobian_cuda<double>(cudafun, x0, real_eval, jc);
    auto t4 = Clock::now();
    auto cuda_time = std::chrono::duration_cast<std::chrono::microseconds>(t4 - t3).count();

    std::cout << "CUDA Jacobian norm:\n" << jc.norm() << std::endl;
    std::cout << "CUDA Time: " << cuda_time << " μs\n\n";

    // Calculate speedup
    double speedup = static_cast<double>(cpu_time) / cuda_time;
    std::cout << "CUDA Speedup: " << speedup << "x\n";
#endif

    return 0;
}